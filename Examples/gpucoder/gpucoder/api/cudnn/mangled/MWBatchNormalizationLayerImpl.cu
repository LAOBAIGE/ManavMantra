#include "MWBatchNormalizationLayer.hpp"
#include "MWBatchNormalizationLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWCNNLayerImpl.hpp"
#include <stdio.h>
#include <cassert> 
 MWBatchNormalizationLayerImpl::MWBatchNormalizationLayerImpl(MWCNNLayer* 
layer, double const MCrRCXUsCsGPMgQbvMOt, const char* 
MEmIeGILUZNEWEagSzRk, const char* MIBnYCbKBdUrlfqlHdoo, const char* 
MNuwXDSoGEYeABeVTwOh, const char* MUmglsoWcEiRiAZsclur, 
MWTargetNetworkImpl* ntwk_impl, int inPlace) : MWCNNLayerImpl(layer, ntwk_impl) 
 , oYbqYsqgVhrUzFEKbBbR(NULL) , jscBrjkVJyVfMMDjFpgl(NULL) , 
ujSEtllBwMdSJhSkFCia(NULL) , vFNECEAeLZsYsUxvlgqL(NULL) , 
aLsOwwcceEmRSYzllBNs(inPlace) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&NtWaRGCHLeTapjWdEHHS)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createBatchNormalizationLayer(MCrRCXUsCsGPMgQbvMOt, MEmIeGILUZNEWEagSzRk, 
MIBnYCbKBdUrlfqlHdoo, MNuwXDSoGEYeABeVTwOh, 
MUmglsoWcEiRiAZsclur); } 
MWBatchNormalizationLayerImpl::~MWBatchNormalizationLayerImpl() { } void 
MWBatchNormalizationLayerImpl::createBatchNormalizationLayer(double const 
MCrRCXUsCsGPMgQbvMOt, const char* MEmIeGILUZNEWEagSzRk, const char* 
MIBnYCbKBdUrlfqlHdoo, const char* MNuwXDSoGEYeABeVTwOh, const char* 
MUmglsoWcEiRiAZsclur) { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* ipTensor = 
BNLayer->getInputTensor(); MWTensor* opTensor = BNLayer->getOutputTensor(); 
UEESbUvbMihFnquvuFij = MCrRCXUsCsGPMgQbvMOt; 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); const 
size_t numBytes = sizeof(float)*ipTensor->getChannels(); 
CUDA_CALL(hipMalloc((void**)&oYbqYsqgVhrUzFEKbBbR, numBytes)); 
CUDA_CALL(hipMalloc((void**)&jscBrjkVJyVfMMDjFpgl, numBytes)); 
CUDA_CALL(hipMalloc((void**)&ujSEtllBwMdSJhSkFCia, numBytes)); 
CUDA_CALL(hipMalloc((void**)&vFNECEAeLZsYsUxvlgqL, numBytes)); 
fvTCtkwXgyScJYogJVFU = HIPDNN_BATCHNORM_SPATIAL; 
CUDNN_CALL(hipdnnDeriveBNTensorDescriptor(NtWaRGCHLeTapjWdEHHS, 
*getOutputDescriptor(), fvTCtkwXgyScJYogJVFU));  loadScale(MIBnYCbKBdUrlfqlHdoo); 
loadOffset(MEmIeGILUZNEWEagSzRk); 
loadTrainedMean(MNuwXDSoGEYeABeVTwOh); 
loadTrainedVariance(MUmglsoWcEiRiAZsclur); if (aLsOwwcceEmRSYzllBNs) 
{ REXdEoRjxuQJkqgIDihy = getLayer()->getInputTensor()->getData(); } else { 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*opTensor->getHeight()* 
opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); } } 
void MWBatchNormalizationLayerImpl::iLoadParamOntoGPU(char const * const 
UzaGmBLFEwmwaFXebUma, int const hnewnpwgzKmOdualajhn, float* 
XCLDbxHBtWRStETWIkId) { FILE* WIxRBCJtmETvfxpuRuus = fopen(UzaGmBLFEwmwaFXebUma,"rb"); 
assert(WIxRBCJtmETvfxpuRuus); int const OVOphSOolqRQDDoKPwxy = 
sizeof(float)*hnewnpwgzKmOdualajhn; float* OKaRVOctKLlnIyGmjRNW = 
(float*)malloc(OVOphSOolqRQDDoKPwxy); fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), 
hnewnpwgzKmOdualajhn, WIxRBCJtmETvfxpuRuus); fclose(WIxRBCJtmETvfxpuRuus); 
CUDA_CALL(hipMemcpy(XCLDbxHBtWRStETWIkId, OKaRVOctKLlnIyGmjRNW, 
OVOphSOolqRQDDoKPwxy, hipMemcpyHostToDevice)); free(OKaRVOctKLlnIyGmjRNW); } 
void MWBatchNormalizationLayerImpl::loadScale(const char* UzaGmBLFEwmwaFXebUma) 
{ MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(UzaGmBLFEwmwaFXebUma, 
opTensor->getChannels(), oYbqYsqgVhrUzFEKbBbR); } void 
MWBatchNormalizationLayerImpl::loadOffset(const char* UzaGmBLFEwmwaFXebUma) { 
MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(UzaGmBLFEwmwaFXebUma, 
opTensor->getChannels(), jscBrjkVJyVfMMDjFpgl); } void 
MWBatchNormalizationLayerImpl::loadTrainedMean(const char* UzaGmBLFEwmwaFXebUma) 
{ MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(UzaGmBLFEwmwaFXebUma, 
opTensor->getChannels(), ujSEtllBwMdSJhSkFCia); } void 
MWBatchNormalizationLayerImpl::loadTrainedVariance(const char* 
UzaGmBLFEwmwaFXebUma) { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(UzaGmBLFEwmwaFXebUma, 
opTensor->getChannels(), vFNECEAeLZsYsUxvlgqL); } void 
MWBatchNormalizationLayerImpl::predict() { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* ipTensor = 
BNLayer->getInputTensor(); MWTensor* opTensor = BNLayer->getOutputTensor(); 
const hipdnnTensorDescriptor_t ZinudJuZuGitiNTsJpBR = 
*getCuDNNDescriptor(ipTensor); float* bDTIjtxZiSHtjwzgEluE = ipTensor->getData(); 
hipdnnTensorDescriptor_t kNsviQGMPdXzNMRixGWR = *getOutputDescriptor(); float* 
kkqTyvjYvRFtTOyQUwrF = getData(); 
CUDNN_CALL(hipdnnBatchNormalizationForwardInference( 
*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), fvTCtkwXgyScJYogJVFU, getOnePtr(), 
getZeroPtr(),  ZinudJuZuGitiNTsJpBR, bDTIjtxZiSHtjwzgEluE, kNsviQGMPdXzNMRixGWR, 
kkqTyvjYvRFtTOyQUwrF, NtWaRGCHLeTapjWdEHHS, oYbqYsqgVhrUzFEKbBbR, 
jscBrjkVJyVfMMDjFpgl, ujSEtllBwMdSJhSkFCia, vFNECEAeLZsYsUxvlgqL, 
UEESbUvbMihFnquvuFij)); } void MWBatchNormalizationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(NtWaRGCHLeTapjWdEHHS)); if 
(hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } 
if(oYbqYsqgVhrUzFEKbBbR) { call_cuda_free(oYbqYsqgVhrUzFEKbBbR); } 
if(jscBrjkVJyVfMMDjFpgl) { call_cuda_free(jscBrjkVJyVfMMDjFpgl); } 
if(ujSEtllBwMdSJhSkFCia) { call_cuda_free(ujSEtllBwMdSJhSkFCia); } 
if(vFNECEAeLZsYsUxvlgqL) { call_cuda_free(vFNECEAeLZsYsUxvlgqL); } 
if (!aLsOwwcceEmRSYzllBNs) { MWTensor* op = getLayer()->getOutputTensor(0); 
float* data = op->getData(); if (data) { call_cuda_free(data); } }  }