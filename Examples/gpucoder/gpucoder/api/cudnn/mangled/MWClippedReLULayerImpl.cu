#include "hip/hip_runtime.h"
#include "MWClippedReLULayerImpl.hpp"
#include "cnn_api.hpp"
#include <math.h>
 MWClippedReLULayerImpl::MWClippedReLULayerImpl(MWCNNLayer* layer , double 
KCudOrFMfgCzUPMcdePX, MWTargetNetworkImpl* ntwk_impl, int inPlace) : 
MWCNNLayerImpl(layer, ntwk_impl) , aLsOwwcceEmRSYzllBNs(inPlace)  { 
createClippedReLULayer(KCudOrFMfgCzUPMcdePX); } void __global__ 
ClippedReLUImpl(float* juRPduBvIGpwaZiftkzr, const double ATEikvMQPqBefhJzjzhc, const int 
CGbFsczkgkhjcHoCKzBx) { int const i = blockDim.x * blockIdx.x + threadIdx.x; if (i < 
CGbFsczkgkhjcHoCKzBx) { float tf = float(juRPduBvIGpwaZiftkzr[i] > 0); juRPduBvIGpwaZiftkzr[i] = 
tf*((juRPduBvIGpwaZiftkzr[i] < ATEikvMQPqBefhJzjzhc) ? juRPduBvIGpwaZiftkzr[i] : ATEikvMQPqBefhJzjzhc); } } 
void MWClippedReLULayerImpl::clippedReLUForwardImpl(int ZCArwzdUdwQuFQUWjnUE, int 
vxtNGOWYjhKeBBSzuIMB, int jLyhrFjMmVnNjoeDJCwH, int NMMfJylfQjiIUAKhXCJb, 
const double OwenhowBxTAXHXmJpIKd, float* output) { int hljcfGWsvZXJZNrImpJB = 
ZCArwzdUdwQuFQUWjnUE*vxtNGOWYjhKeBBSzuIMB* 
jLyhrFjMmVnNjoeDJCwH*NMMfJylfQjiIUAKhXCJb; int omxlPZbBePZdWaJOBUUG = 
std::floor(static_cast<float>(hljcfGWsvZXJZNrImpJB)/static_cast<float>(32)) * 32; 
int sRECVoNNtDdcBOWgDyar = (omxlPZbBePZdWaJOBUUG < 1024) ? omxlPZbBePZdWaJOBUUG : 
1024; int NnAKUXChhnRnQmWsknGy = (hljcfGWsvZXJZNrImpJB + 
sRECVoNNtDdcBOWgDyar - 1)/sRECVoNNtDdcBOWgDyar; 
ClippedReLUImpl<<<NnAKUXChhnRnQmWsknGy, 
sRECVoNNtDdcBOWgDyar>>>(output, OwenhowBxTAXHXmJpIKd, hljcfGWsvZXJZNrImpJB); 
} void MWClippedReLULayerImpl::createClippedReLULayer(double 
KCudOrFMfgCzUPMcdePX) { OwenhowBxTAXHXmJpIKd = KCudOrFMfgCzUPMcdePX; MWTensor* op = 
getLayer()->getOutputTensor(); if (aLsOwwcceEmRSYzllBNs) { REXdEoRjxuQJkqgIDihy = 
getLayer()->getInputTensor()->getData(); } else { 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, sizeof(float)*op->getHeight()* 
op->getWidth()*op->getChannels()*op->getBatchSize())); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, op->getBatchSize(), op->getChannels(), 
op->getHeight(), op->getWidth())); } void MWClippedReLULayerImpl::predict() { 
MWTensor* op = getLayer()->getOutputTensor(0); 
clippedReLUForwardImpl(op->getHeight(), op->getWidth(), op->getChannels(), 
op->getBatchSize(), OwenhowBxTAXHXmJpIKd, getData()); } void 
MWClippedReLULayerImpl::cleanup() { if (hasOutputDescriptor()) {  
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } if 
(!aLsOwwcceEmRSYzllBNs) { MWTensor* op = getLayer()->getOutputTensor(0); float* 
data = op->getData(); if (data) { call_cuda_free(data); } }  }