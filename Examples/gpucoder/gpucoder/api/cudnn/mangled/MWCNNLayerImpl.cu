#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t WprSrhAStKGxyXeoxETy; void 
curand_call_line_file(hiprandStatus_t olKGEIcsxmLSoMhRhEtP, const int 
fOpFYwKNwIfWjnPzNuob, const char *UKtMXCCqdjeyaVHabkxg) { if (olKGEIcsxmLSoMhRhEtP != 
HIPRAND_STATUS_SUCCESS) { printf("%d, line: %d, file: %s\n", olKGEIcsxmLSoMhRhEtP, 
fOpFYwKNwIfWjnPzNuob, UKtMXCCqdjeyaVHabkxg); exit(EXIT_FAILURE); } }
#endif
 float* malloc_call_line_file(size_t msize, const int fOpFYwKNwIfWjnPzNuob, const 
char *UKtMXCCqdjeyaVHabkxg) { float * mem = (float*)malloc(msize); if (!mem) { 
printf("%s, line: %d, file: %s\n", "Memory allocation failed. ", 
fOpFYwKNwIfWjnPzNuob, UKtMXCCqdjeyaVHabkxg); exit(EXIT_FAILURE); } return mem; } void 
call_cuda_free(float* mem) { hipError_t olKGEIcsxmLSoMhRhEtP = hipFree(mem); if 
(olKGEIcsxmLSoMhRhEtP != hipErrorDeinitialized) { CUDA_CALL(olKGEIcsxmLSoMhRhEtP); 
} } void cuda_call_line_file(hipError_t olKGEIcsxmLSoMhRhEtP, const int 
fOpFYwKNwIfWjnPzNuob, const char *UKtMXCCqdjeyaVHabkxg) { if (olKGEIcsxmLSoMhRhEtP != 
hipSuccess) { printf("%s, line: %d, file: %s\n", 
hipGetErrorString(olKGEIcsxmLSoMhRhEtP), fOpFYwKNwIfWjnPzNuob, UKtMXCCqdjeyaVHabkxg); 
exit(EXIT_FAILURE); } } void cudnn_call_line_file(hipdnnStatus_t 
olKGEIcsxmLSoMhRhEtP, const int fOpFYwKNwIfWjnPzNuob, const char *UKtMXCCqdjeyaVHabkxg) { if 
(olKGEIcsxmLSoMhRhEtP != HIPDNN_STATUS_SUCCESS) { 
printf("%s, line: %d, file: %s\n", hipdnnGetErrorString(olKGEIcsxmLSoMhRhEtP), 
fOpFYwKNwIfWjnPzNuob, UKtMXCCqdjeyaVHabkxg); exit(EXIT_FAILURE); } } const char* 
cublasGetErrorString(hipblasStatus_t olKGEIcsxmLSoMhRhEtP) { 
switch(olKGEIcsxmLSoMhRhEtP) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t olKGEIcsxmLSoMhRhEtP, const int 
fOpFYwKNwIfWjnPzNuob, const char *UKtMXCCqdjeyaVHabkxg) { if (olKGEIcsxmLSoMhRhEtP != 
HIPBLAS_STATUS_SUCCESS) { printf("%s, line: %d, file: %s\n", 
cublasGetErrorString(olKGEIcsxmLSoMhRhEtP), fOpFYwKNwIfWjnPzNuob, UKtMXCCqdjeyaVHabkxg); 
exit(EXIT_FAILURE); } } MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl) : TxNFOfYScyqGlEFFxbAv(0.0), SGsAudmgjmvcUXzzrUtf(1.0), 
SDWKEQTZaTFZByPlzUDR(-1.0), eybNKlJCSDUvsznWynwK(layer), 
gzSTokDHvkXefhiGDcWL(ntwk_impl), REXdEoRjxuQJkqgIDihy(0)  { } float* 
MWCNNLayerImpl::getZeroPtr() { return &TxNFOfYScyqGlEFFxbAv; } float* 
MWCNNLayerImpl::getOnePtr() { return &SGsAudmgjmvcUXzzrUtf; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &SDWKEQTZaTFZByPlzUDR; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getOutputDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lWJYwWaFPmWNQDPrlqER.find(index); if (it == lWJYwWaFPmWNQDPrlqER.end()) { 
hipdnnTensorDescriptor_t* tmp = new hipdnnTensorDescriptor_t; if (!tmp) { 
printf("%s, line: %d, file: %s\n", 
"Error! Out of memory. Unable to allocate output descriptors. ", __LINE__ , 
__FILE__); exit(EXIT_FAILURE); } lWJYwWaFPmWNQDPrlqER[index] = tmp; return 
tmp; } else { assert(it->second); return it->second; } } bool 
MWCNNLayerImpl::hasOutputDescriptor(int index) const { std::map<int, 
hipdnnTensorDescriptor_t*>::const_iterator it = 
lWJYwWaFPmWNQDPrlqER.find(index); return (it != lWJYwWaFPmWNQDPrlqER.end()); 
} hipdnnTensorDescriptor_t* MWCNNLayerImpl::getCuDNNDescriptor(MWTensor* tensor) 
{ MWCNNLayerImpl* impl = tensor->getOwner()->getImpl(); if (!impl || 
dynamic_cast<MWPassthroughLayer*>(tensor->getOwner())) { 
assert(dynamic_cast<MWPassthroughLayer*>(tensor->getOwner())); return 
getCuDNNDescriptor(tensor->getOwner()->getInputTensor(0)); } return 
impl->getOutputDescriptor(tensor->getSourcePortIndex()); } 
MWInputLayerImpl::MWInputLayerImpl(MWCNNLayer* layer, int fxxCPKTclxXPxrdMAkwi, int 
YgcpEBUCwCLaPhyntIio, int vIWQzNvYZSuxmOTVDFhU, int OumvfgWXDdmsQaciHMHx, bool wMySyzzledUmSLTWhuYH, 
const char* avg_file_name, MWTargetNetworkImpl* ntwk_impl) : 
MWCNNLayerImpl(layer, ntwk_impl) { createInputLayer(fxxCPKTclxXPxrdMAkwi, YgcpEBUCwCLaPhyntIio, 
vIWQzNvYZSuxmOTVDFhU, OumvfgWXDdmsQaciHMHx, wMySyzzledUmSLTWhuYH, avg_file_name); } 
MWInputLayerImpl::~MWInputLayerImpl() { } void 
MWInputLayerImpl::createInputLayer(int fxxCPKTclxXPxrdMAkwi, int YgcpEBUCwCLaPhyntIio, int 
vIWQzNvYZSuxmOTVDFhU, int OumvfgWXDdmsQaciHMHx, bool wMySyzzledUmSLTWhuYH, const char* 
avg_file_name){ CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU*OumvfgWXDdmsQaciHMHx*fxxCPKTclxXPxrdMAkwi)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MdSWZSOAjugbWppryHbR)); 
euppfEoiaoCTcVgRPVhA = wMySyzzledUmSLTWhuYH; 
gzSTokDHvkXefhiGDcWL->setWorkSpaceSize(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fxxCPKTclxXPxrdMAkwi, OumvfgWXDdmsQaciHMHx, YgcpEBUCwCLaPhyntIio, 
vIWQzNvYZSuxmOTVDFhU)); if( euppfEoiaoCTcVgRPVhA ) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MdSWZSOAjugbWppryHbR, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, OumvfgWXDdmsQaciHMHx, YgcpEBUCwCLaPhyntIio, vIWQzNvYZSuxmOTVDFhU)); 
CUDA_CALL(hipMalloc((void**)&JwxFdqOKggeawILBfGgg, 
sizeof(float)*OumvfgWXDdmsQaciHMHx*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU)); int hDaNSVZAofAENeIAiWEw = 
OumvfgWXDdmsQaciHMHx*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU;  loadAvg(avg_file_name, 
hDaNSVZAofAENeIAiWEw); }
#ifdef RANDOM
 hiprandGenerateUniform(WprSrhAStKGxyXeoxETy, MW_data, fxxCPKTclxXPxrdMAkwi*OumvfgWXDdmsQaciHMHx*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU);
#endif
 gzSTokDHvkXefhiGDcWL->setWorkSpaceSize(0); return; } void 
MWInputLayerImpl::loadAvg(const char* UdmcwaUkepxfZrpdpcAN, int hDaNSVZAofAENeIAiWEw) 
{ FILE* WIxRBCJtmETvfxpuRuus = MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN); 
assert(WIxRBCJtmETvfxpuRuus); float* OKaRVOctKLlnIyGmjRNW = 
MALLOC_CALL(sizeof(float)*hDaNSVZAofAENeIAiWEw); fread(OKaRVOctKLlnIyGmjRNW, 
sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); 
CUDA_CALL(hipMemcpy(JwxFdqOKggeawILBfGgg, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; } void 
MWInputLayerImpl::predict() { if ( euppfEoiaoCTcVgRPVhA ) 
CUDNN_CALL(hipdnnAddTensor(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
getNegOnePtr(), MdSWZSOAjugbWppryHbR, JwxFdqOKggeawILBfGgg, getOnePtr(), 
*getOutputDescriptor(), getData())); return; } void MWInputLayerImpl::cleanup() 
{ if (hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } for(int idx 
= 0; idx < eybNKlJCSDUvsznWynwK->getNumOutputs(); idx++) {  float* data = 
eybNKlJCSDUvsznWynwK->getOutputTensor(idx)->getData(); if (data) { 
call_cuda_free(data); } } if ( euppfEoiaoCTcVgRPVhA ) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MdSWZSOAjugbWppryHbR)); if (JwxFdqOKggeawILBfGgg) 
{ call_cuda_free(JwxFdqOKggeawILBfGgg); } } return; } 
MWConvLayerImpl::MWConvLayerImpl(MWCNNLayer* layer, int filt_H, int filt_W, int 
numGrps, int numChnls, int numFilts, int IbSWJNMuIiKbocfQKqXb, int 
IwKnaBoXVubIRYcxEJLH, int GeeOVBfQrpMacIFBLKOo, int GFienSVKLlDQuZeqAdLC, int 
GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, const char* 
vjDFlBZzKvbpPseAtMBP, const char* NldNILHvuQqQPSAHXxdT, MWTargetNetworkImpl* 
ntwk_impl)  : MWCNNLayerImpl(layer, ntwk_impl)  , xkUNToJIgvoLoUQuzKRF(NULL) , 
vIWQzNvYZSuxmOTVDFhU(NULL) , NDjzAZSYJuWymuKDNZYB(NULL) , veFyKKHbdqBIvQLYBqfF(NULL) , 
ZDWLzHUkuZuIUZHfbGDY(NULL) , dJcdBfQQLhIAYHPxwQeg(NULL) , eqOmMKQRpqBqRQCnJmxt(0) , 
AwZQzUhuWVLGrWgLHRuM(filt_H) , AzTsxYcYjIEJsGQbeYHm (filt_W) , 
DSsxcjIrUgZCKZovyNQf (numGrps) , CZNYmBcNFSZWvaCklqeM (numChnls) , 
CpMjJjtGOeWOzwxpAAQP (numFilts) { gzSTokDHvkXefhiGDcWL = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&QMgBqCuvjnbWHWiVPEwn)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&VCbcPxtPsBLTrHYdEvqn)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&NZjOkZPwLzQsdEVkwMcX)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createConvLayer(IbSWJNMuIiKbocfQKqXb, IwKnaBoXVubIRYcxEJLH, GeeOVBfQrpMacIFBLKOo, 
GFienSVKLlDQuZeqAdLC, GsZlHFuhbvjLtRMDjXnW, HJHXkKmgFxxIOsIvRRnF, 
vjDFlBZzKvbpPseAtMBP, NldNILHvuQqQPSAHXxdT); } 
MWConvLayerImpl::~MWConvLayerImpl() { } float MWConvLayerImpl::getIsGrouped() { 
return eqOmMKQRpqBqRQCnJmxt; } void MWConvLayerImpl::setIsGrouped(float ig) { 
eqOmMKQRpqBqRQCnJmxt = ig; return; } void MWConvLayerImpl::setOutput2(float* 
out2) { xkUNToJIgvoLoUQuzKRF = out2; return; } float* MWConvLayerImpl::getOutput2() { 
return xkUNToJIgvoLoUQuzKRF; } hipdnnTensorDescriptor_t* 
MWConvLayerImpl::getGroupDescriptor() { return &XVcMnvCXvZpKICKIjgZi; } void 
MWConvLayerImpl::createConvLayer(int IbSWJNMuIiKbocfQKqXb, int 
IwKnaBoXVubIRYcxEJLH, int GeeOVBfQrpMacIFBLKOo, int GFienSVKLlDQuZeqAdLC , int 
GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, const char* 
vjDFlBZzKvbpPseAtMBP, const char* NldNILHvuQqQPSAHXxdT) { MWTensor* ipTensor 
= getLayer()->getInputTensor(0); int QVgVGfoCXYiYXzPhvVPX = 
GeeOVBfQrpMacIFBLKOo; int QhTesEEIHwhNmHSeYbRR = 
GsZlHFuhbvjLtRMDjXnW; if ((GeeOVBfQrpMacIFBLKOo != GFienSVKLlDQuZeqAdLC) || 
(GsZlHFuhbvjLtRMDjXnW != HJHXkKmgFxxIOsIvRRnF)) { float* newInput; int inputH 
= ipTensor->getHeight() + GeeOVBfQrpMacIFBLKOo + GFienSVKLlDQuZeqAdLC; int 
inputW = ipTensor->getWidth() + GsZlHFuhbvjLtRMDjXnW + HJHXkKmgFxxIOsIvRRnF; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*ipTensor->getBatchSize() 
* ipTensor->getChannels() * inputH * inputW)); CUDA_CALL(hipMemset(newInput, 
0, 
sizeof(float)*ipTensor->getBatchSize()*ipTensor->getChannels()*inputH*inputW)); 
ZDWLzHUkuZuIUZHfbGDY = new MWTensor(inputH, inputW, ipTensor->getChannels(), 
ipTensor->getBatchSize(), newInput,getLayer(), 0); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&eFaDPmxDdzHlRYSAoMmX)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(eFaDPmxDdzHlRYSAoMmX, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, ZDWLzHUkuZuIUZHfbGDY->getBatchSize(), ZDWLzHUkuZuIUZHfbGDY->getChannels(), 
ZDWLzHUkuZuIUZHfbGDY->getHeight(), ZDWLzHUkuZuIUZHfbGDY->getWidth())); 
QVgVGfoCXYiYXzPhvVPX = 0;  QhTesEEIHwhNmHSeYbRR = 0;  } else { 
ZDWLzHUkuZuIUZHfbGDY = ipTensor; eFaDPmxDdzHlRYSAoMmX = 
*getCuDNNDescriptor(ZDWLzHUkuZuIUZHfbGDY);  } fSKMHAqIghbYYgyIpNDw = 
GeeOVBfQrpMacIFBLKOo; fhikqqlnUKCjleVKDqiG = GsZlHFuhbvjLtRMDjXnW;  
assert(ZDWLzHUkuZuIUZHfbGDY != NULL); MWConvLayer* convLayer = static_cast<MWConvLayer*>(getLayer());
#if (CUDNN_MAJOR <= 5)
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(QMgBqCuvjnbWHWiVPEwn, 
QVgVGfoCXYiYXzPhvVPX, QhTesEEIHwhNmHSeYbRR, IbSWJNMuIiKbocfQKqXb, 
IwKnaBoXVubIRYcxEJLH, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(QMgBqCuvjnbWHWiVPEwn, 
QVgVGfoCXYiYXzPhvVPX, QhTesEEIHwhNmHSeYbRR, IbSWJNMuIiKbocfQKqXb, 
IwKnaBoXVubIRYcxEJLH, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));  }
#endif
 int qWwjVYwfnvEnFKlgpqwA, pckLLTEdVPoCZLRwyDnM; int numInputFeatures = 
CZNYmBcNFSZWvaCklqeM*DSsxcjIrUgZCKZovyNQf; int 
jhFUWlztBndwjbXwYNaJ,puSFZkRJmyuFPfQRswDK,rSmEWccbJFfPGddhPemm; MWTensor* 
opTensor = convLayer->getOutputTensor(0); jhFUWlztBndwjbXwYNaJ = 
opTensor->getChannels(); puSFZkRJmyuFPfQRswDK = opTensor->getHeight(); 
rSmEWccbJFfPGddhPemm = opTensor->getWidth();  size_t sxuOMwKXOKfuExclRaSe = 0; if( 
DSsxcjIrUgZCKZovyNQf == 1 ) { 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(VCbcPxtPsBLTrHYdEvqn, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, jhFUWlztBndwjbXwYNaJ, numInputFeatures, 
AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm));  
CUDNN_CALL(hipdnnSetTensor4dDescriptor(NZjOkZPwLzQsdEVkwMcX, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, jhFUWlztBndwjbXwYNaJ, 1, 1)); 
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(QMgBqCuvjnbWHWiVPEwn, 
eFaDPmxDdzHlRYSAoMmX, VCbcPxtPsBLTrHYdEvqn, &qWwjVYwfnvEnFKlgpqwA, 
&pckLLTEdVPoCZLRwyDnM, &puSFZkRJmyuFPfQRswDK, &rSmEWccbJFfPGddhPemm)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, qWwjVYwfnvEnFKlgpqwA, pckLLTEdVPoCZLRwyDnM, 
opTensor->getHeight(), opTensor->getWidth())); assert(opTensor->getHeight() == 
puSFZkRJmyuFPfQRswDK); assert(opTensor->getWidth() == rSmEWccbJFfPGddhPemm);
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
eFaDPmxDdzHlRYSAoMmX, VCbcPxtPsBLTrHYdEvqn, QMgBqCuvjnbWHWiVPEwn, 
*getOutputDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, 
&PmFfARVzoHVAYkfpuvqK)); }
#else
 { hipdnnConvolutionFwdAlgoPerf_t perf_results[3]; int returnedAlgoCount; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
eFaDPmxDdzHlRYSAoMmX, VCbcPxtPsBLTrHYdEvqn, QMgBqCuvjnbWHWiVPEwn, 
*getOutputDescriptor(), 3, &returnedAlgoCount, perf_results)); 
PmFfARVzoHVAYkfpuvqK = perf_results[0].algo; }
#endif
 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
eFaDPmxDdzHlRYSAoMmX, VCbcPxtPsBLTrHYdEvqn, QMgBqCuvjnbWHWiVPEwn, 
*getOutputDescriptor(), PmFfARVzoHVAYkfpuvqK, &sxuOMwKXOKfuExclRaSe)); } else { 
setIsGrouped(1); MWTensor* ipTensor = ZDWLzHUkuZuIUZHfbGDY;  dJcdBfQQLhIAYHPxwQeg = 
ipTensor->getData() + ipTensor->getChannels()/DSsxcjIrUgZCKZovyNQf * 
ipTensor->getHeight() * ipTensor->getWidth(); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&enPbWLzEmxYCBmzGJutZ)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(enPbWLzEmxYCBmzGJutZ, 
HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(), 
ipTensor->getChannels()/DSsxcjIrUgZCKZovyNQf, ipTensor->getHeight(), 
ipTensor->getWidth(), 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth(), 
ipTensor->getHeight()*ipTensor->getWidth(), ipTensor->getWidth(), 1)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getGroupDescriptor()));  
CUDNN_CALL(hipdnnSetFilter4dDescriptor(VCbcPxtPsBLTrHYdEvqn, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, CpMjJjtGOeWOzwxpAAQP, CZNYmBcNFSZWvaCklqeM, 
AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm));  
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(QMgBqCuvjnbWHWiVPEwn, 
enPbWLzEmxYCBmzGJutZ, VCbcPxtPsBLTrHYdEvqn, &qWwjVYwfnvEnFKlgpqwA, 
&pckLLTEdVPoCZLRwyDnM, &puSFZkRJmyuFPfQRswDK, &rSmEWccbJFfPGddhPemm)); 
assert(opTensor->getHeight() == puSFZkRJmyuFPfQRswDK); assert(opTensor->getWidth() 
== rSmEWccbJFfPGddhPemm); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(*getGroupDescriptor(), 
HIPDNN_DATA_FLOAT, qWwjVYwfnvEnFKlgpqwA, pckLLTEdVPoCZLRwyDnM, puSFZkRJmyuFPfQRswDK, 
rSmEWccbJFfPGddhPemm, 
pckLLTEdVPoCZLRwyDnM*DSsxcjIrUgZCKZovyNQf*puSFZkRJmyuFPfQRswDK*rSmEWccbJFfPGddhPemm, 
puSFZkRJmyuFPfQRswDK*rSmEWccbJFfPGddhPemm, rSmEWccbJFfPGddhPemm, 1)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, qWwjVYwfnvEnFKlgpqwA, 
pckLLTEdVPoCZLRwyDnM*DSsxcjIrUgZCKZovyNQf, puSFZkRJmyuFPfQRswDK, rSmEWccbJFfPGddhPemm)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(NZjOkZPwLzQsdEVkwMcX, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, pckLLTEdVPoCZLRwyDnM*DSsxcjIrUgZCKZovyNQf, 1, 1));
#if (CUDNN_MAJOR < 7) 
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
enPbWLzEmxYCBmzGJutZ, VCbcPxtPsBLTrHYdEvqn, QMgBqCuvjnbWHWiVPEwn, 
*getGroupDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &PmFfARVzoHVAYkfpuvqK));
#else
 hipdnnConvolutionFwdAlgoPerf_t perf_results[3]; int returnedAlgoCount; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
enPbWLzEmxYCBmzGJutZ, VCbcPxtPsBLTrHYdEvqn, QMgBqCuvjnbWHWiVPEwn, 
*getGroupDescriptor(), 3, &returnedAlgoCount,perf_results)); 
PmFfARVzoHVAYkfpuvqK = perf_results[0].algo;
#endif
 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
enPbWLzEmxYCBmzGJutZ, VCbcPxtPsBLTrHYdEvqn, QMgBqCuvjnbWHWiVPEwn, 
*getGroupDescriptor(), PmFfARVzoHVAYkfpuvqK, &sxuOMwKXOKfuExclRaSe)); } if( 
sxuOMwKXOKfuExclRaSe > *gzSTokDHvkXefhiGDcWL->getWorkSpaceSize() ) { 
gzSTokDHvkXefhiGDcWL->setWorkSpaceSize(sxuOMwKXOKfuExclRaSe); }  
assert(qWwjVYwfnvEnFKlgpqwA == ipTensor->getBatchSize()); 
assert(jhFUWlztBndwjbXwYNaJ == pckLLTEdVPoCZLRwyDnM * 
DSsxcjIrUgZCKZovyNQf); CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float) * opTensor->getBatchSize() * opTensor->getChannels() * 
opTensor->getHeight() * opTensor->getWidth())); 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float)*CZNYmBcNFSZWvaCklqeM*jhFUWlztBndwjbXwYNaJ*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm)); 
CUDA_CALL(hipMalloc((void**)&NDjzAZSYJuWymuKDNZYB, sizeof(float)*jhFUWlztBndwjbXwYNaJ));
#ifdef RANDOM
 hiprandGenerateNormal(WprSrhAStKGxyXeoxETy, vIWQzNvYZSuxmOTVDFhU, 
CZNYmBcNFSZWvaCklqeM*jhFUWlztBndwjbXwYNaJ*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm, 
0, 0.1); hiprandGenerateNormal(WprSrhAStKGxyXeoxETy, NDjzAZSYJuWymuKDNZYB, 
jhFUWlztBndwjbXwYNaJ, -0.5, 1);
#endif
 if( DSsxcjIrUgZCKZovyNQf == 2 ) { veFyKKHbdqBIvQLYBqfF = vIWQzNvYZSuxmOTVDFhU + 
CpMjJjtGOeWOzwxpAAQP * CZNYmBcNFSZWvaCklqeM * AwZQzUhuWVLGrWgLHRuM * 
AzTsxYcYjIEJsGQbeYHm; setOutput2(getData() + jhFUWlztBndwjbXwYNaJ/ 2 
* puSFZkRJmyuFPfQRswDK * rSmEWccbJFfPGddhPemm); setIsGrouped(1); } 
loadWeights(vjDFlBZzKvbpPseAtMBP); loadBias(NldNILHvuQqQPSAHXxdT); return; } 
void __global__ padInputImpl(float* in, int inputH, int inputW, int inputCh, 
int outputH, int outputW, int offsetH, int offsetW, float* out, int inputElems) 
{ for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < inputElems; i+= 
blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); int rem = (i - 
idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); int rem1 = rem 
- idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = rem1 - 
idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = idxH + 
offsetH; int outputCol = idxCol + offsetW; int outputCh = inputCh; *(out + 
idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + outputR*(outputW) + 
outputCol) = *(in + i); } } } void MWConvLayerImpl::predict() { MWConvLayer* 
convLayer = static_cast<MWConvLayer*>(getLayer()); if (ZDWLzHUkuZuIUZHfbGDY != 
convLayer->getInputTensor()) { CUDA_CALL(hipMemset(ZDWLzHUkuZuIUZHfbGDY->getData(), 
0, 
sizeof(float)*ZDWLzHUkuZuIUZHfbGDY->getBatchSize()*ZDWLzHUkuZuIUZHfbGDY->getChannels()*ZDWLzHUkuZuIUZHfbGDY->getHeight()*ZDWLzHUkuZuIUZHfbGDY->getWidth())); 
 int iPqBiFnIJMxelVhQBZex = 
convLayer->getInputTensor()->getHeight()*convLayer->getInputTensor()->getWidth()*convLayer->getInputTensor()->getBatchSize()*convLayer->getInputTensor()->getChannels(); 
int sRECVoNNtDdcBOWgDyar = (iPqBiFnIJMxelVhQBZex < 1024) ? 
iPqBiFnIJMxelVhQBZex : 1024; int NnAKUXChhnRnQmWsknGy = (iPqBiFnIJMxelVhQBZex 
+ sRECVoNNtDdcBOWgDyar - 1)/sRECVoNNtDdcBOWgDyar;  
padInputImpl<<<NnAKUXChhnRnQmWsknGy, 
sRECVoNNtDdcBOWgDyar>>>(convLayer->getInputTensor()->getData(), 
convLayer->getInputTensor()->getHeight(), 
convLayer->getInputTensor()->getWidth(), 
convLayer->getInputTensor()->getChannels(), ZDWLzHUkuZuIUZHfbGDY->getHeight(), 
ZDWLzHUkuZuIUZHfbGDY->getWidth(), fSKMHAqIghbYYgyIpNDw, fhikqqlnUKCjleVKDqiG,  
ZDWLzHUkuZuIUZHfbGDY->getData(), iPqBiFnIJMxelVhQBZex); } if(DSsxcjIrUgZCKZovyNQf == 1 
) { assert(getData() != ZDWLzHUkuZuIUZHfbGDY->getData()); 
CUDNN_CALL(hipdnnConvolutionForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(),getOnePtr(), 
eFaDPmxDdzHlRYSAoMmX, ZDWLzHUkuZuIUZHfbGDY->getData(), VCbcPxtPsBLTrHYdEvqn, 
vIWQzNvYZSuxmOTVDFhU, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK, 
gzSTokDHvkXefhiGDcWL->getWorkSpace(), *gzSTokDHvkXefhiGDcWL->getWorkSpaceSize(), 
getZeroPtr(), *getOutputDescriptor(),getData())); 
CUDNN_CALL(hipdnnAddTensor(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), getOnePtr(), 
NZjOkZPwLzQsdEVkwMcX, NDjzAZSYJuWymuKDNZYB, getOnePtr(), 
*getOutputDescriptor(),getData())); } else { assert(getData() != 
ZDWLzHUkuZuIUZHfbGDY->getData()); 
CUDNN_CALL(hipdnnConvolutionForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
getOnePtr(), enPbWLzEmxYCBmzGJutZ, ZDWLzHUkuZuIUZHfbGDY->getData(), 
VCbcPxtPsBLTrHYdEvqn, vIWQzNvYZSuxmOTVDFhU, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK, 
gzSTokDHvkXefhiGDcWL->getWorkSpace(), *gzSTokDHvkXefhiGDcWL->getWorkSpaceSize(), 
getZeroPtr(), *getGroupDescriptor(), getData())); 
CUDNN_CALL(hipdnnConvolutionForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
getOnePtr(), enPbWLzEmxYCBmzGJutZ, dJcdBfQQLhIAYHPxwQeg, VCbcPxtPsBLTrHYdEvqn, 
veFyKKHbdqBIvQLYBqfF, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK, 
gzSTokDHvkXefhiGDcWL->getWorkSpace(), *gzSTokDHvkXefhiGDcWL->getWorkSpaceSize(), 
getZeroPtr(), *getGroupDescriptor(), getOutput2())); 
CUDNN_CALL(hipdnnAddTensor(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), getOnePtr(), 
NZjOkZPwLzQsdEVkwMcX, NDjzAZSYJuWymuKDNZYB, getOnePtr(), *getOutputDescriptor(), 
getData())); } } void MWConvLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(QMgBqCuvjnbWHWiVPEwn)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(VCbcPxtPsBLTrHYdEvqn)); if 
(vIWQzNvYZSuxmOTVDFhU) { call_cuda_free(vIWQzNvYZSuxmOTVDFhU); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(NZjOkZPwLzQsdEVkwMcX)); if 
(NDjzAZSYJuWymuKDNZYB) { call_cuda_free(NDjzAZSYJuWymuKDNZYB); } if (hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } if 
(ZDWLzHUkuZuIUZHfbGDY != getLayer()->getInputTensor(0)) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(eFaDPmxDdzHlRYSAoMmX)); 
call_cuda_free(ZDWLzHUkuZuIUZHfbGDY->getData()); } if (getIsGrouped()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(enPbWLzEmxYCBmzGJutZ));  
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getGroupDescriptor())); } for(int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { call_cuda_free(data); 
} } return; } void MWConvLayerImpl::loadWeights(const char* 
UdmcwaUkepxfZrpdpcAN) { MWConvLayer* convLayer = 
static_cast<MWConvLayer*>(getLayer()); FILE* WIxRBCJtmETvfxpuRuus = 
MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN); assert(WIxRBCJtmETvfxpuRuus); 
assert(CZNYmBcNFSZWvaCklqeM == 
ZDWLzHUkuZuIUZHfbGDY->getChannels()/DSsxcjIrUgZCKZovyNQf); int hDaNSVZAofAENeIAiWEw = 
CZNYmBcNFSZWvaCklqeM*convLayer->getOutputTensor()->getChannels()*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm; 
 float* OKaRVOctKLlnIyGmjRNW = MALLOC_CALL(sizeof(float)*hDaNSVZAofAENeIAiWEw); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); if( 
AwZQzUhuWVLGrWgLHRuM != 1 && AzTsxYcYjIEJsGQbeYHm != 1 ) { float* 
ONvcEjLBnVNUdjMKOAwF = 
MALLOC_CALL(sizeof(float)*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm); 
for(int k=0; k<hDaNSVZAofAENeIAiWEw/AwZQzUhuWVLGrWgLHRuM/AzTsxYcYjIEJsGQbeYHm; 
k++) { for(int i=0; i<AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm; i++) 
ONvcEjLBnVNUdjMKOAwF[i]=OKaRVOctKLlnIyGmjRNW[k*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm+i]; 
for(int j=0; j<AwZQzUhuWVLGrWgLHRuM; j++) for(int i=0; 
i<AzTsxYcYjIEJsGQbeYHm; i++) 
OKaRVOctKLlnIyGmjRNW[k*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm+j*AzTsxYcYjIEJsGQbeYHm+i]=ONvcEjLBnVNUdjMKOAwF[j+i*AwZQzUhuWVLGrWgLHRuM]; 
} free(ONvcEjLBnVNUdjMKOAwF); } CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
OKaRVOctKLlnIyGmjRNW, sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", UdmcwaUkepxfZrpdpcAN, hDaNSVZAofAENeIAiWEw, OKaRVOctKLlnIyGmjRNW[0]);
#endif
 free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; } void 
MWConvLayerImpl::loadBias(const char* UdmcwaUkepxfZrpdpcAN) { MWConvLayer* 
convLayer = static_cast<MWConvLayer*>(getLayer()); FILE* WIxRBCJtmETvfxpuRuus = 
MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN);  assert(WIxRBCJtmETvfxpuRuus); int 
hDaNSVZAofAENeIAiWEw = convLayer->getOutputTensor()->getChannels();  float* 
OKaRVOctKLlnIyGmjRNW = MALLOC_CALL(sizeof(float)*hDaNSVZAofAENeIAiWEw); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); 
CUDA_CALL(hipMemcpy(NDjzAZSYJuWymuKDNZYB, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; } 
MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int inPlace)  : MWCNNLayerImpl(layer, ntwk_impl) , 
aLsOwwcceEmRSYzllBNs(inPlace)  { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&npGnQZLrEfVTQnEbwqij)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createReLULayer(); } MWReLULayerImpl::~MWReLULayerImpl() { } void 
MWReLULayerImpl::createReLULayer() { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); MWTensor* ipTensor = 
reluLayer->getInputTensor(0); MWTensor* opTensor = 
reluLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetActivationDescriptor(npGnQZLrEfVTQnEbwqij, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));  
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); if 
(aLsOwwcceEmRSYzllBNs) {  REXdEoRjxuQJkqgIDihy = 
reluLayer->getInputTensor()->getData(); } else { 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*opTensor->getHeight()* 
opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); }  } 
void MWReLULayerImpl::predict() { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(reluLayer->getInputTensor()); 
CUDNN_CALL(hipdnnActivationForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
npGnQZLrEfVTQnEbwqij, getOnePtr(), ipDesc, 
reluLayer->getInputTensor()->getData(), getZeroPtr(), *getOutputDescriptor(), 
REXdEoRjxuQJkqgIDihy)); } void MWReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(npGnQZLrEfVTQnEbwqij)); if 
(hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } if 
(!aLsOwwcceEmRSYzllBNs) { MWTensor* op = getLayer()->getOutputTensor(0); float* 
data = op->getData(); if (data) { call_cuda_free(data); } }  } 
MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, unsigned 
JgLfgHrHMEMmMYTettJF,  double AHqhysOOIgbDpWZoPUFT,  double 
AIXLuRgdeiqpaCehGSYD,  double BRSPqxNffoBYKqpSVHne, MWTargetNetworkImpl* ntwk_impl) : 
MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateLRNDescriptor(&gTcJMwtYuwiqqUmqvKhT)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createNormLayer(JgLfgHrHMEMmMYTettJF, AHqhysOOIgbDpWZoPUFT, 
AIXLuRgdeiqpaCehGSYD, BRSPqxNffoBYKqpSVHne); } MWNormLayerImpl::~MWNormLayerImpl() { } void 
MWNormLayerImpl::createNormLayer( unsigned JgLfgHrHMEMmMYTettJF,  
double AHqhysOOIgbDpWZoPUFT,  double AIXLuRgdeiqpaCehGSYD,  double BRSPqxNffoBYKqpSVHne) { 
MWNormLayer* normLayer = static_cast<MWNormLayer*>(getLayer()); MWTensor* 
ipTensor = normLayer->getInputTensor(0); MWTensor* opTensor = 
normLayer->getOutputTensor(0); int numOutputFeatures = opTensor->getChannels(); 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*opTensor->getHeight()*opTensor->getWidth()*numOutputFeatures*opTensor->getBatchSize())); 
CUDNN_CALL(hipdnnSetLRNDescriptor(gTcJMwtYuwiqqUmqvKhT, 
JgLfgHrHMEMmMYTettJF, AHqhysOOIgbDpWZoPUFT, AIXLuRgdeiqpaCehGSYD, 
BRSPqxNffoBYKqpSVHne)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); return; 
} void MWNormLayerImpl::predict() { MWNormLayer* normLayer = 
static_cast<MWNormLayer*>(getLayer()); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(normLayer->getInputTensor()); 
CUDNN_CALL(hipdnnLRNCrossChannelForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
gTcJMwtYuwiqqUmqvKhT, HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), ipDesc, 
normLayer->getInputTensor()->getData(),getZeroPtr(), *getOutputDescriptor(), 
normLayer->getOutputTensor()->getData())); } void MWNormLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyLRNDescriptor(gTcJMwtYuwiqqUmqvKhT)); if 
(hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } for(int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) {  MWTensor* op = 
getLayer()->getOutputTensor(idx); float* data = op->getData(); if (data) { 
call_cuda_free(data); } }  } void __global__ MWSetDyForBackPropImpl(float * 
SIBpKtDURUWQaaenbwrC, const int jaqKGCwoANNDMHgAsehk); void __global__ 
doMWMaxPoolingLayerImpl(float * cQBKlCKXxecGPJrXBXdk, float * 
cCXqPFPPcoHzYMDpnUxQ, const int CGbFsczkgkhjcHoCKzBx); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, int 
HtQBsWTCGEkpylRklilw,  int IAlDgIFcchbwRGBSfVfA,  int IbSWJNMuIiKbocfQKqXb,  int 
IwKnaBoXVubIRYcxEJLH, int GeeOVBfQrpMacIFBLKOo, int GFienSVKLlDQuZeqAdLC,  int 
GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, bool KHClOltUSuqFVVErSxVb, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) , 
BLjrjqvCcCommiXWQLjs(KHClOltUSuqFVVErSxVb) , cQBKlCKXxecGPJrXBXdk(0) 
, SIBpKtDURUWQaaenbwrC(0) , cCXqPFPPcoHzYMDpnUxQ(0)  {  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&lteHjcLsItGbVPMQtGDB)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createMaxPoolingLayer(HtQBsWTCGEkpylRklilw,IAlDgIFcchbwRGBSfVfA,IbSWJNMuIiKbocfQKqXb,IwKnaBoXVubIRYcxEJLH,GeeOVBfQrpMacIFBLKOo,GFienSVKLlDQuZeqAdLC,GsZlHFuhbvjLtRMDjXnW,HJHXkKmgFxxIOsIvRRnF); 
} MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::createMaxPoolingLayer(int HtQBsWTCGEkpylRklilw,  int 
IAlDgIFcchbwRGBSfVfA,  int IbSWJNMuIiKbocfQKqXb, int IwKnaBoXVubIRYcxEJLH, int 
GeeOVBfQrpMacIFBLKOo, int GFienSVKLlDQuZeqAdLC,  int GsZlHFuhbvjLtRMDjXnW, 
int HJHXkKmgFxxIOsIvRRnF) { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
maxpoolLayer->getInputTensor(0); int nNULvWnBXnnWdpEkHPAH = 
GeeOVBfQrpMacIFBLKOo; int nlIRrOJaFuVaywxOqOyb = 
GsZlHFuhbvjLtRMDjXnW; hipdnnTensorDescriptor_t eFaDPmxDdzHlRYSAoMmX = 
*getCuDNNDescriptor(ipTensor);  
CUDNN_CALL(hipdnnSetPooling2dDescriptor(lteHjcLsItGbVPMQtGDB, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, HtQBsWTCGEkpylRklilw, IAlDgIFcchbwRGBSfVfA, 
nNULvWnBXnnWdpEkHPAH, nlIRrOJaFuVaywxOqOyb, IbSWJNMuIiKbocfQKqXb, 
IwKnaBoXVubIRYcxEJLH)); int fxxCPKTclxXPxrdMAkwi, OumvfgWXDdmsQaciHMHx, YgcpEBUCwCLaPhyntIio, 
vIWQzNvYZSuxmOTVDFhU; CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(lteHjcLsItGbVPMQtGDB, 
eFaDPmxDdzHlRYSAoMmX, &fxxCPKTclxXPxrdMAkwi ,&OumvfgWXDdmsQaciHMHx, &YgcpEBUCwCLaPhyntIio, 
&vIWQzNvYZSuxmOTVDFhU)); YgcpEBUCwCLaPhyntIio = getLayer()->getOutputTensor(0)->getHeight(); 
vIWQzNvYZSuxmOTVDFhU = getLayer()->getOutputTensor(0)->getWidth(); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fxxCPKTclxXPxrdMAkwi, OumvfgWXDdmsQaciHMHx, YgcpEBUCwCLaPhyntIio, 
vIWQzNvYZSuxmOTVDFhU)); CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*fxxCPKTclxXPxrdMAkwi*OumvfgWXDdmsQaciHMHx*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU)); if 
(BLjrjqvCcCommiXWQLjs){ 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(1))); const int 
hljcfGWsvZXJZNrImpJB = 
(ipTensor->getHeight())*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
CUDA_CALL(hipMalloc((void**)&cQBKlCKXxecGPJrXBXdk, 
sizeof(float)*hljcfGWsvZXJZNrImpJB)); 
CUDA_CALL(hipMalloc((void**)&cCXqPFPPcoHzYMDpnUxQ, 
sizeof(float)*fxxCPKTclxXPxrdMAkwi*OumvfgWXDdmsQaciHMHx*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU)); 
assert((OumvfgWXDdmsQaciHMHx == ipTensor->getChannels()) && (fxxCPKTclxXPxrdMAkwi == 
ipTensor->getBatchSize()));  const int jaqKGCwoANNDMHgAsehk = 
vIWQzNvYZSuxmOTVDFhU*YgcpEBUCwCLaPhyntIio*OumvfgWXDdmsQaciHMHx*fxxCPKTclxXPxrdMAkwi; 
CUDA_CALL(hipMalloc((void**)&SIBpKtDURUWQaaenbwrC, 
sizeof(float)*jaqKGCwoANNDMHgAsehk)); int sRECVoNNtDdcBOWgDyar = 
(jaqKGCwoANNDMHgAsehk < 1024) ? jaqKGCwoANNDMHgAsehk : 1024; int 
NnAKUXChhnRnQmWsknGy = (jaqKGCwoANNDMHgAsehk + sRECVoNNtDdcBOWgDyar - 
1)/sRECVoNNtDdcBOWgDyar; 
MWSetDyForBackPropImpl<<<NnAKUXChhnRnQmWsknGy, 
sRECVoNNtDdcBOWgDyar>>>( SIBpKtDURUWQaaenbwrC, jaqKGCwoANNDMHgAsehk); } } void 
MWMaxPoolingLayerImpl::predict() { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); hipdnnTensorDescriptor_t 
eFaDPmxDdzHlRYSAoMmX = *getCuDNNDescriptor(maxpoolLayer->getInputTensor()); 
MWTensor* ipTensor = getLayer()->getInputTensor(0); 
CUDNN_CALL(hipdnnPoolingForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
lteHjcLsItGbVPMQtGDB, getOnePtr(), eFaDPmxDdzHlRYSAoMmX, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(), 
maxpoolLayer->getOutputTensor()->getData())); if (BLjrjqvCcCommiXWQLjs) { 
CUDNN_CALL(hipdnnPoolingBackward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
lteHjcLsItGbVPMQtGDB, getOnePtr(), *getOutputDescriptor(0), 
getLayer()->getOutputTensor(0)->getData(), *getOutputDescriptor(0), 
SIBpKtDURUWQaaenbwrC, eFaDPmxDdzHlRYSAoMmX, ipTensor->getData(), getZeroPtr(), 
eFaDPmxDdzHlRYSAoMmX, cQBKlCKXxecGPJrXBXdk)); int hljcfGWsvZXJZNrImpJB = 
ipTensor->getHeight()*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
int sRECVoNNtDdcBOWgDyar = (hljcfGWsvZXJZNrImpJB < 1024) ? 
hljcfGWsvZXJZNrImpJB : 1024; int NnAKUXChhnRnQmWsknGy = (hljcfGWsvZXJZNrImpJB + 
sRECVoNNtDdcBOWgDyar - 1)/sRECVoNNtDdcBOWgDyar; 
doMWMaxPoolingLayerImpl<<<NnAKUXChhnRnQmWsknGy, 
sRECVoNNtDdcBOWgDyar>>>( cQBKlCKXxecGPJrXBXdk, 
maxpoolLayer->getOutputTensor(1)->getData(), hljcfGWsvZXJZNrImpJB); } return; } 
void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(lteHjcLsItGbVPMQtGDB)); if 
(hasOutputDescriptor(0)) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(0))); } if 
(BLjrjqvCcCommiXWQLjs) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(1))); } for(int 
idx = 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { call_cuda_free(data); 
} } if (cQBKlCKXxecGPJrXBXdk){ 
call_cuda_free(cQBKlCKXxecGPJrXBXdk); } if (SIBpKtDURUWQaaenbwrC){ 
call_cuda_free(SIBpKtDURUWQaaenbwrC); }  } float* 
MWMaxPoolingLayerImpl::getIndexData()  { return cCXqPFPPcoHzYMDpnUxQ; } void 
__global__ MWSetDyForBackPropImpl(float * SIBpKtDURUWQaaenbwrC, const int 
jaqKGCwoANNDMHgAsehk) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
jaqKGCwoANNDMHgAsehk; i+= blockDim.x*gridDim.x) { SIBpKtDURUWQaaenbwrC[i] = i+1; } } 
void __global__ doMWMaxPoolingLayerImpl(float * cQBKlCKXxecGPJrXBXdk, 
float * cCXqPFPPcoHzYMDpnUxQ, const int CGbFsczkgkhjcHoCKzBx) { for(int i = 
blockDim.x * blockIdx.x + threadIdx.x; i < CGbFsczkgkhjcHoCKzBx; i+= 
blockDim.x*gridDim.x) { if (static_cast<int>(cQBKlCKXxecGPJrXBXdk[i]) 
!= 0){ 
cCXqPFPPcoHzYMDpnUxQ[static_cast<int>(cQBKlCKXxecGPJrXBXdk[i])-1] = 
i; } } } MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, const char* 
vjDFlBZzKvbpPseAtMBP,  const char* NldNILHvuQqQPSAHXxdT, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&NZjOkZPwLzQsdEVkwMcX)); 
createFCLayer(vjDFlBZzKvbpPseAtMBP, NldNILHvuQqQPSAHXxdT); } 
MWFCLayerImpl::~MWFCLayerImpl() { } void MWFCLayerImpl::createFCLayer(const 
char* vjDFlBZzKvbpPseAtMBP, const char* NldNILHvuQqQPSAHXxdT) { MWFCLayer* 
fcLayer = static_cast<MWFCLayer*>(getLayer()); 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*fcLayer->getOutputTensor()->getBatchSize()*fcLayer->getOutputTensor()->getChannels())); 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float)*fcLayer->getInputTensor()->getChannels() 
*fcLayer->getInputTensor()->getWidth()*fcLayer->getInputTensor()->getHeight()*fcLayer->getOutputTensor()->getChannels())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 
fcLayer->getOutputTensor()->getBatchSize(),fcLayer->getOutputTensor()->getChannels(), 
1, 1)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(NZjOkZPwLzQsdEVkwMcX, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 
fcLayer->getOutputTensor()->getChannels(), 1, 1)); 
CUDA_CALL(hipMalloc((void**)&NDjzAZSYJuWymuKDNZYB, sizeof(float)*fcLayer->getOutputTensor()->getChannels()));
#ifdef RANDOM
 hiprandGenerateNormal(WprSrhAStKGxyXeoxETy, vIWQzNvYZSuxmOTVDFhU, 
fcLayer->getInputTensor()->getChannels()*fcLayer->getInputTensor()->getWidth()*fcLayer->getInputTensor()->getHeight()*fcLayer->getOutputTensor()->getChannels(), 
0, 0.1); hiprandGenerateNormal(WprSrhAStKGxyXeoxETy, NDjzAZSYJuWymuKDNZYB, 
fcLayer->getOutputTensor()->getChannels(), -0.5, 1);
#endif
 loadWeights(vjDFlBZzKvbpPseAtMBP); loadBias(NldNILHvuQqQPSAHXxdT); return; 
} void MWFCLayerImpl::loadWeights(const char* UdmcwaUkepxfZrpdpcAN) {  
MWFCLayer* fcLayer = static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); MWTensor* opTensor = fcLayer->getOutputTensor(0); 
FILE* WIxRBCJtmETvfxpuRuus = MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN); 
assert(WIxRBCJtmETvfxpuRuus); int hDaNSVZAofAENeIAiWEw = 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth()*opTensor->getChannels(); 
 float* OKaRVOctKLlnIyGmjRNW = MALLOC_CALL(sizeof(float)*hDaNSVZAofAENeIAiWEw); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); if( 
ipTensor->getHeight() != 1 && ipTensor->getWidth() != 1 ) { float* 
ONvcEjLBnVNUdjMKOAwF = 
MALLOC_CALL(sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()); for(int 
k=0; k<hDaNSVZAofAENeIAiWEw/ipTensor->getHeight()/ipTensor->getWidth(); k++) { 
for(int i=0; i<ipTensor->getHeight()*ipTensor->getWidth(); i++) 
ONvcEjLBnVNUdjMKOAwF[i]=OKaRVOctKLlnIyGmjRNW[k*ipTensor->getHeight()*ipTensor->getWidth()+i]; 
for(int j=0; j<ipTensor->getHeight(); j++) for(int i=0; i<ipTensor->getWidth(); 
i++) 
OKaRVOctKLlnIyGmjRNW[k*ipTensor->getHeight()*ipTensor->getWidth()+j*ipTensor->getWidth()+i]=ONvcEjLBnVNUdjMKOAwF[j+i*ipTensor->getHeight()]; 
} free(ONvcEjLBnVNUdjMKOAwF); } CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
OKaRVOctKLlnIyGmjRNW, sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", UdmcwaUkepxfZrpdpcAN, hDaNSVZAofAENeIAiWEw, OKaRVOctKLlnIyGmjRNW[0]);
#endif
 free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; } void 
MWFCLayerImpl::loadBias(const char* UdmcwaUkepxfZrpdpcAN) { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* opTensor = 
fcLayer->getOutputTensor(0); FILE* WIxRBCJtmETvfxpuRuus = 
MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN); assert(WIxRBCJtmETvfxpuRuus); int 
hDaNSVZAofAENeIAiWEw = opTensor->getChannels();  float* OKaRVOctKLlnIyGmjRNW = 
MALLOC_CALL(sizeof(float)*hDaNSVZAofAENeIAiWEw); fread(OKaRVOctKLlnIyGmjRNW, 
sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); 
CUDA_CALL(hipMemcpy(NDjzAZSYJuWymuKDNZYB, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; } void 
MWFCLayerImpl::predict() { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); MWTensor* opTensor = fcLayer->getOutputTensor(0); 
int DqxLTLaJwwgQqmrtCDuu = 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth(); int 
ECTnqgWHyHCHCLBZlffd = opTensor->getChannels(); int bMAyVFGSPDjmUbziYLAy=1; 
int bUVPfnrJhLfHzOLUUrKk=1; if( opTensor->getBatchSize()==1 ) { 
CUDA_CALL(hipMemcpy(getData(), NDjzAZSYJuWymuKDNZYB, 
sizeof(float)*ECTnqgWHyHCHCLBZlffd, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*gzSTokDHvkXefhiGDcWL->getCublasHandle(), HIPBLAS_OP_T, 
DqxLTLaJwwgQqmrtCDuu, ECTnqgWHyHCHCLBZlffd, getOnePtr(), 
vIWQzNvYZSuxmOTVDFhU, DqxLTLaJwwgQqmrtCDuu, ipTensor->getData(), 
bMAyVFGSPDjmUbziYLAy, getOnePtr(),getData(), bUVPfnrJhLfHzOLUUrKk)); } else { 
CUBLAS_CALL(hipblasSgemm(*gzSTokDHvkXefhiGDcWL->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, ECTnqgWHyHCHCLBZlffd, opTensor->getBatchSize(), 
DqxLTLaJwwgQqmrtCDuu, getOnePtr(), vIWQzNvYZSuxmOTVDFhU, 
DqxLTLaJwwgQqmrtCDuu, ipTensor->getData(), DqxLTLaJwwgQqmrtCDuu, 
getZeroPtr(),getData(), ECTnqgWHyHCHCLBZlffd)); 
CUDNN_CALL(hipdnnAddTensor(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), getOnePtr(), 
NZjOkZPwLzQsdEVkwMcX, NDjzAZSYJuWymuKDNZYB, getOnePtr(), 
*getOutputDescriptor(),getData())); } return; } void MWFCLayerImpl::cleanup() { 
if (vIWQzNvYZSuxmOTVDFhU) { call_cuda_free(vIWQzNvYZSuxmOTVDFhU); } if (hasOutputDescriptor(0)) 
{ CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(0))); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(NZjOkZPwLzQsdEVkwMcX)); if 
(NDjzAZSYJuWymuKDNZYB) { call_cuda_free(NDjzAZSYJuWymuKDNZYB); } for(int idx = 0; idx < 
getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { call_cuda_free(data); 
} } } MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  {  
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createSoftmaxLayer(); } MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() { } void 
MWSoftmaxLayerImpl::createSoftmaxLayer() { MWSoftmaxLayer* sfmxLayer = 
static_cast<MWSoftmaxLayer*>(getLayer()); MWTensor* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensor* opTensor = 
sfmxLayer->getOutputTensor(0); int numOutputFeatures = ipTensor->getChannels(); 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()*numOutputFeatures*ipTensor->getBatchSize())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth()));  
return; } void MWSoftmaxLayerImpl::predict() { MWSoftmaxLayer* sfmxLayer = 
static_cast<MWSoftmaxLayer*>(getLayer()); MWTensor* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensor* opTensor = 
sfmxLayer->getOutputTensor(0); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(ipTensor);  
CUDNN_CALL(hipdnnSoftmaxForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), ipDesc, 
ipTensor->getData(), getZeroPtr(), *getOutputDescriptor(), getData())); } void 
MWSoftmaxLayerImpl::cleanup() { if (hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } for(int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { call_cuda_free(data); 
} } } MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, int 
HtQBsWTCGEkpylRklilw,  int IAlDgIFcchbwRGBSfVfA,  int IbSWJNMuIiKbocfQKqXb,  int 
IwKnaBoXVubIRYcxEJLH,  int FrpxvsDMwwgbpqHXWxmN,  int GnxRkpzrPZimKtYYHSuG, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&lteHjcLsItGbVPMQtGDB)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createAvgPoolingLayer(HtQBsWTCGEkpylRklilw, IAlDgIFcchbwRGBSfVfA, IbSWJNMuIiKbocfQKqXb, 
IwKnaBoXVubIRYcxEJLH, FrpxvsDMwwgbpqHXWxmN, GnxRkpzrPZimKtYYHSuG); } 
MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() { } void 
MWAvgPoolingLayerImpl::createAvgPoolingLayer(int HtQBsWTCGEkpylRklilw, int 
IAlDgIFcchbwRGBSfVfA, int IbSWJNMuIiKbocfQKqXb, int IwKnaBoXVubIRYcxEJLH, int 
FrpxvsDMwwgbpqHXWxmN, int GnxRkpzrPZimKtYYHSuG) { MWAvgPoolingLayer* avgpoolLayer 
= static_cast<MWAvgPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
avgpoolLayer->getInputTensor(0); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(lteHjcLsItGbVPMQtGDB, 
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, HIPDNN_NOT_PROPAGATE_NAN, 
HtQBsWTCGEkpylRklilw, IAlDgIFcchbwRGBSfVfA, FrpxvsDMwwgbpqHXWxmN, GnxRkpzrPZimKtYYHSuG, 
IbSWJNMuIiKbocfQKqXb, IwKnaBoXVubIRYcxEJLH)); int fxxCPKTclxXPxrdMAkwi, OumvfgWXDdmsQaciHMHx, 
YgcpEBUCwCLaPhyntIio, vIWQzNvYZSuxmOTVDFhU;  hipdnnTensorDescriptor_t eFaDPmxDdzHlRYSAoMmX = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(lteHjcLsItGbVPMQtGDB, 
eFaDPmxDdzHlRYSAoMmX, &fxxCPKTclxXPxrdMAkwi ,&OumvfgWXDdmsQaciHMHx, &YgcpEBUCwCLaPhyntIio, 
&vIWQzNvYZSuxmOTVDFhU)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, fxxCPKTclxXPxrdMAkwi, OumvfgWXDdmsQaciHMHx, YgcpEBUCwCLaPhyntIio, 
vIWQzNvYZSuxmOTVDFhU)); CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*fxxCPKTclxXPxrdMAkwi*OumvfgWXDdmsQaciHMHx*YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU)); } void 
MWAvgPoolingLayerImpl::predict() { MWAvgPoolingLayer* avgpoolLayer = 
static_cast<MWAvgPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
avgpoolLayer->getInputTensor(0); MWTensor* opTensor = 
avgpoolLayer->getOutputTensor(0); hipdnnTensorDescriptor_t eFaDPmxDdzHlRYSAoMmX = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnPoolingForward(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
lteHjcLsItGbVPMQtGDB, getOnePtr(), eFaDPmxDdzHlRYSAoMmX, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(),opTensor->getData())); } void 
MWAvgPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(lteHjcLsItGbVPMQtGDB)); if 
(hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); }  for(int 
idx = 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { call_cuda_free(data); 
} } } MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { 
createOutputLayer(); } MWOutputLayerImpl::~MWOutputLayerImpl() { } void 
MWOutputLayerImpl::createOutputLayer() { MWOutputLayer* opLayer = 
static_cast<MWOutputLayer*>(getLayer()); MWTensor* ipTensor = 
opLayer->getInputTensor(0); setData(ipTensor->getData()); return; } void 
MWOutputLayerImpl::predict() { } void MWOutputLayerImpl::cleanup() { }