#include "hip/hip_runtime.h"
#include "MWMaxUnpoolingLayerImpl.hpp"
#include "MWMaxUnpoolingLayer.hpp"
#include <cassert>
 MWMaxUnpoolingLayerImpl::MWMaxUnpoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { 
createUnpoolingLayer(); } MWMaxUnpoolingLayerImpl::~MWMaxUnpoolingLayerImpl() { 
 } void MWMaxUnpoolingLayerImpl::createUnpoolingLayer() { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*opTensor->getBatchSize()* opTensor->getChannels()* 
opTensor->getHeight()* opTensor->getWidth())); 
CUDA_CALL(hipMemset(REXdEoRjxuQJkqgIDihy,0.0f, 
sizeof(float)*opTensor->getBatchSize()* opTensor->getChannels()* 
opTensor->getHeight()* opTensor->getWidth() ));  
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
MWMaxUnpoolingLayerImpl::predict() { assert(this->getData() != 
getLayer()->getInputTensor(0)->getData()); 
doMaxUnpoolingForwardImpl(getLayer()->getInputTensor(0)->getData(), 
getLayer()->getInputTensor(1)->getData(), 
getLayer()->getOutputTensor(0)->getData(), 
getLayer()->getInputTensor(0)->getHeight(), 
getLayer()->getInputTensor(0)->getWidth(), 
getLayer()->getInputTensor(0)->getChannels(), 
getLayer()->getInputTensor(0)->getBatchSize()); return; } void __global__ 
MaxUnpoolingImpl(float * inputBuffer, float * indexBuffer, float * 
outputBuffer, const int CGbFsczkgkhjcHoCKzBx) { for(int i = blockDim.x * blockIdx.x + 
threadIdx.x; i < CGbFsczkgkhjcHoCKzBx; i+= blockDim.x*gridDim.x) { 
outputBuffer[static_cast<int>(indexBuffer[i])] = inputBuffer[i]; } } void 
MWMaxUnpoolingLayerImpl::doMaxUnpoolingForwardImpl(float* inputBuffer, float* 
indexBuffer, float* outputBuffer, int ZCArwzdUdwQuFQUWjnUE, int vxtNGOWYjhKeBBSzuIMB, 
int jLyhrFjMmVnNjoeDJCwH, int NMMfJylfQjiIUAKhXCJb ) { int 
hljcfGWsvZXJZNrImpJB = ZCArwzdUdwQuFQUWjnUE*vxtNGOWYjhKeBBSzuIMB* 
jLyhrFjMmVnNjoeDJCwH*NMMfJylfQjiIUAKhXCJb; int 
sRECVoNNtDdcBOWgDyar = (hljcfGWsvZXJZNrImpJB < 1024) ? hljcfGWsvZXJZNrImpJB : 
1024; int NnAKUXChhnRnQmWsknGy = (hljcfGWsvZXJZNrImpJB + 
sRECVoNNtDdcBOWgDyar - 1)/sRECVoNNtDdcBOWgDyar; 
MaxUnpoolingImpl<<<NnAKUXChhnRnQmWsknGy, sRECVoNNtDdcBOWgDyar>>>( 
inputBuffer, indexBuffer, outputBuffer, hljcfGWsvZXJZNrImpJB); } void 
MWMaxUnpoolingLayerImpl::cleanup() { if (hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } for(int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) { float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { call_cuda_free(data); 
} } }