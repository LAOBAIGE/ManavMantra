#include <stdio.h>
#include "cnn_api.hpp"
#include "MWTransposedConvolution2DLayer.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWTransposedConvolution2DLayerImpl.hpp"
#include <cassert>
 
MWTransposedConvolution2DLayerImpl::MWTransposedConvolution2DLayerImpl(MWCNNLayer* 
layer, int filt_H, int filt_W, int numIpFeatures, int numFilts, int 
IbSWJNMuIiKbocfQKqXb, int IwKnaBoXVubIRYcxEJLH, int FrpxvsDMwwgbpqHXWxmN, int 
GnxRkpzrPZimKtYYHSuG,  const char* vjDFlBZzKvbpPseAtMBP, const char* 
NldNILHvuQqQPSAHXxdT, MWTargetNetworkImpl* ntwk_impl)  : MWCNNLayerImpl(layer, 
ntwk_impl)  , vIWQzNvYZSuxmOTVDFhU(NULL) , NDjzAZSYJuWymuKDNZYB(NULL) , 
AwZQzUhuWVLGrWgLHRuM(filt_H) , AzTsxYcYjIEJsGQbeYHm(filt_W) , 
DqxLTLaJwwgQqmrtCDuu(numIpFeatures) , CpMjJjtGOeWOzwxpAAQP(numFilts) { 
gzSTokDHvkXefhiGDcWL = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&RqCYCrGsNvzKYrRMXbsI)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&VCbcPxtPsBLTrHYdEvqn)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&NZjOkZPwLzQsdEVkwMcX)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createTransposedConv2DLayer(IbSWJNMuIiKbocfQKqXb, IwKnaBoXVubIRYcxEJLH, 
FrpxvsDMwwgbpqHXWxmN, GnxRkpzrPZimKtYYHSuG, vjDFlBZzKvbpPseAtMBP, 
NldNILHvuQqQPSAHXxdT); } 
MWTransposedConvolution2DLayerImpl::~MWTransposedConvolution2DLayerImpl() { } 
void MWTransposedConvolution2DLayerImpl::createTransposedConv2DLayer(int 
IbSWJNMuIiKbocfQKqXb, int IwKnaBoXVubIRYcxEJLH, int FrpxvsDMwwgbpqHXWxmN, int 
GnxRkpzrPZimKtYYHSuG, const char* vjDFlBZzKvbpPseAtMBP, const char* 
NldNILHvuQqQPSAHXxdT) { MWTransposedConvolution2DLayer* convLayer = static_cast<MWTransposedConvolution2DLayer*>(getLayer());
#if (CUDNN_MAJOR <= 5)
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(RqCYCrGsNvzKYrRMXbsI, 
FrpxvsDMwwgbpqHXWxmN, GnxRkpzrPZimKtYYHSuG, IbSWJNMuIiKbocfQKqXb, 
IwKnaBoXVubIRYcxEJLH, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(RqCYCrGsNvzKYrRMXbsI, 
FrpxvsDMwwgbpqHXWxmN, GnxRkpzrPZimKtYYHSuG, IbSWJNMuIiKbocfQKqXb, 
IwKnaBoXVubIRYcxEJLH, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));  }
#endif
 int numOutFeatures_fwdConv = DqxLTLaJwwgQqmrtCDuu;  int 
numInFeatures_fwdConv = CpMjJjtGOeWOzwxpAAQP;  
CUDNN_CALL(hipdnnSetFilter4dDescriptor(VCbcPxtPsBLTrHYdEvqn, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, numOutFeatures_fwdConv, numInFeatures_fwdConv, 
AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm));  MWTensor* ipTensor = 
convLayer->getInputTensor(0); MWTensor* opTensor = 
convLayer->getOutputTensor(0); int puSFZkRJmyuFPfQRswDK = opTensor->getHeight(); 
int rSmEWccbJFfPGddhPemm = opTensor->getWidth(); hipdnnTensorDescriptor_t 
eFaDPmxDdzHlRYSAoMmX = *getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), puSFZkRJmyuFPfQRswDK, rSmEWccbJFfPGddhPemm)); 
CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
VCbcPxtPsBLTrHYdEvqn,*getCuDNNDescriptor(ipTensor), RqCYCrGsNvzKYrRMXbsI, 
*getOutputDescriptor(), HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, 
&PtkeOkuClHzhOfpmBevf)); size_t ugnnrhsgTeWucrMPCJUc = 0; 
CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
VCbcPxtPsBLTrHYdEvqn,*getCuDNNDescriptor(ipTensor), RqCYCrGsNvzKYrRMXbsI, 
*getOutputDescriptor(), PtkeOkuClHzhOfpmBevf, &ugnnrhsgTeWucrMPCJUc)); 
if( ugnnrhsgTeWucrMPCJUc > *gzSTokDHvkXefhiGDcWL->getWorkSpaceSize()) { 
gzSTokDHvkXefhiGDcWL->setWorkSpaceSize(ugnnrhsgTeWucrMPCJUc); } 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float)*DqxLTLaJwwgQqmrtCDuu*opTensor->getChannels()*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(NZjOkZPwLzQsdEVkwMcX, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, opTensor->getChannels(), 1, 1)); 
CUDA_CALL(hipMalloc((void**)&NDjzAZSYJuWymuKDNZYB, 
sizeof(float)*opTensor->getChannels())); 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, sizeof(float) * 
opTensor->getBatchSize() * opTensor->getChannels() * opTensor->getHeight() * 
opTensor->getWidth())); loadWeights(vjDFlBZzKvbpPseAtMBP); 
loadBias(NldNILHvuQqQPSAHXxdT); return; } void 
MWTransposedConvolution2DLayerImpl::predict() { MWTransposedConvolution2DLayer* 
convLayer = static_cast<MWTransposedConvolution2DLayer*>(getLayer()); MWTensor* 
ipTensor = convLayer->getInputTensor(0); MWTensor* opTensor = 
convLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnConvolutionBackwardData(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), 
getOnePtr(), VCbcPxtPsBLTrHYdEvqn, vIWQzNvYZSuxmOTVDFhU,  
*getCuDNNDescriptor(ipTensor), ipTensor->getData(), RqCYCrGsNvzKYrRMXbsI,  
PtkeOkuClHzhOfpmBevf, gzSTokDHvkXefhiGDcWL->getWorkSpace(),  
*gzSTokDHvkXefhiGDcWL->getWorkSpaceSize(), getZeroPtr(),  
*getOutputDescriptor(), opTensor->getData())); 
CUDNN_CALL(hipdnnAddTensor(*gzSTokDHvkXefhiGDcWL->getCudnnHandle(), getOnePtr(),  
NZjOkZPwLzQsdEVkwMcX, NDjzAZSYJuWymuKDNZYB, getOnePtr(), 
*getOutputDescriptor(),opTensor->getData())); } void 
MWTransposedConvolution2DLayerImpl::cleanup(){ 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(RqCYCrGsNvzKYrRMXbsI)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(VCbcPxtPsBLTrHYdEvqn)); if 
(vIWQzNvYZSuxmOTVDFhU) { call_cuda_free(vIWQzNvYZSuxmOTVDFhU); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(NZjOkZPwLzQsdEVkwMcX)); if 
(NDjzAZSYJuWymuKDNZYB) { call_cuda_free(NDjzAZSYJuWymuKDNZYB); } if (hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } if 
(getData()) { call_cuda_free(getData()); } } void 
MWTransposedConvolution2DLayerImpl::loadWeights(const char* 
UdmcwaUkepxfZrpdpcAN) { MWTransposedConvolution2DLayer* convLayer = 
static_cast<MWTransposedConvolution2DLayer*>(getLayer()); MWTensor* opTensor = 
convLayer->getOutputTensor(0); FILE* WIxRBCJtmETvfxpuRuus = 
MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN); assert(WIxRBCJtmETvfxpuRuus); int 
hDaNSVZAofAENeIAiWEw = 
DqxLTLaJwwgQqmrtCDuu*opTensor->getChannels()*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm; 
 float* OKaRVOctKLlnIyGmjRNW = (float*)malloc(sizeof(float)*hDaNSVZAofAENeIAiWEw); 
fread(OKaRVOctKLlnIyGmjRNW, sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); if( 
AwZQzUhuWVLGrWgLHRuM != 1 && AzTsxYcYjIEJsGQbeYHm != 1 ) { float* 
ONvcEjLBnVNUdjMKOAwF = 
(float*)malloc(sizeof(float)*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm); 
for(int k=0; k<hDaNSVZAofAENeIAiWEw/AwZQzUhuWVLGrWgLHRuM/AzTsxYcYjIEJsGQbeYHm; 
k++) { for(int i=0; i<AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm; i++) 
ONvcEjLBnVNUdjMKOAwF[i]=OKaRVOctKLlnIyGmjRNW[k*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm+i]; 
for(int j=0; j<AwZQzUhuWVLGrWgLHRuM; j++) for(int i=0; 
i<AzTsxYcYjIEJsGQbeYHm; i++) 
OKaRVOctKLlnIyGmjRNW[k*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm+j*AzTsxYcYjIEJsGQbeYHm+i]=ONvcEjLBnVNUdjMKOAwF[j+i*AwZQzUhuWVLGrWgLHRuM]; 
} free(ONvcEjLBnVNUdjMKOAwF); } CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
OKaRVOctKLlnIyGmjRNW, sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice)); 
printf("%s loaded. Size = %d. %f\n", UdmcwaUkepxfZrpdpcAN, hDaNSVZAofAENeIAiWEw, 
OKaRVOctKLlnIyGmjRNW[0]); free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; 
} void MWTransposedConvolution2DLayerImpl::loadBias(const char* 
UdmcwaUkepxfZrpdpcAN) { MWTransposedConvolution2DLayer* convLayer = 
static_cast<MWTransposedConvolution2DLayer*>(getLayer()); MWTensor* opTensor = 
convLayer->getOutputTensor(0); FILE* WIxRBCJtmETvfxpuRuus = 
MWCNNLayer::openBinaryFile(UdmcwaUkepxfZrpdpcAN); assert(WIxRBCJtmETvfxpuRuus); int 
hDaNSVZAofAENeIAiWEw = opTensor->getChannels();  float* OKaRVOctKLlnIyGmjRNW = 
(float*)malloc(sizeof(float)*hDaNSVZAofAENeIAiWEw); fread(OKaRVOctKLlnIyGmjRNW, 
sizeof(float), hDaNSVZAofAENeIAiWEw, WIxRBCJtmETvfxpuRuus); 
CUDA_CALL(hipMemcpy(NDjzAZSYJuWymuKDNZYB, OKaRVOctKLlnIyGmjRNW, 
sizeof(float)*hDaNSVZAofAENeIAiWEw, hipMemcpyHostToDevice)); 
free(OKaRVOctKLlnIyGmjRNW); fclose(WIxRBCJtmETvfxpuRuus); return; }