#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWCNNLayerImpl.hpp"
 void MWTargetNetworkImpl::preSetup() { QjgQHaUACFNSteMrRtRj = new 
hipblasHandle_t; hipblasCreate(QjgQHaUACFNSteMrRtRj); 
QwUuNuQNtlPXrIwRNiSZ = new hipdnnHandle_t; 
hipdnnCreate(QwUuNuQNtlPXrIwRNiSZ); } void MWTargetNetworkImpl::postSetup() 
{ createWorkSpace(&wtNPjzxHKNoJIigzXrEl); } void 
MWTargetNetworkImpl::createWorkSpace(float** xHViLEwTujGGrPZZgmbF) { 
CUDA_CALL(hipMalloc((void**)xHViLEwTujGGrPZZgmbF, 
oJUVMnJggjhEdQLWzIUC)); } void 
MWTargetNetworkImpl::setWorkSpaceSize(size_t wss) { oJUVMnJggjhEdQLWzIUC 
= wss;  } size_t* MWTargetNetworkImpl::getWorkSpaceSize() { return 
&oJUVMnJggjhEdQLWzIUC; } float* MWTargetNetworkImpl::getWorkSpace() { 
return wtNPjzxHKNoJIigzXrEl; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return QjgQHaUACFNSteMrRtRj; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
QwUuNuQNtlPXrIwRNiSZ; } void MWTargetNetworkImpl::cleanup() { if 
(wtNPjzxHKNoJIigzXrEl) { hipFree(wtNPjzxHKNoJIigzXrEl); } if 
(QjgQHaUACFNSteMrRtRj) { hipblasDestroy(*QjgQHaUACFNSteMrRtRj); } if 
(QwUuNuQNtlPXrIwRNiSZ) { hipdnnDestroy(*QwUuNuQNtlPXrIwRNiSZ); } }