#include "hip/hip_runtime.h"
#include "MWLeakyReLULayerImpl.hpp"
#include "MWLeakyReLULayer.hpp"
 MWLeakyReLULayerImpl::MWLeakyReLULayerImpl(MWCNNLayer* layer, double 
LtEgcYoEYjkrWuohutgw, MWTargetNetworkImpl* ntwk_impl, int inPlace) : 
MWCNNLayerImpl(layer, ntwk_impl) , oYbqYsqgVhrUzFEKbBbR(LtEgcYoEYjkrWuohutgw) , 
aLsOwwcceEmRSYzllBNs(inPlace) { MWTensor* op = getLayer()->getOutputTensor(); if 
(inPlace) { REXdEoRjxuQJkqgIDihy = getLayer()->getInputTensor()->getData(); } else { 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, sizeof(float)*op->getHeight()* 
op->getWidth()*op->getChannels()*op->getBatchSize())); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, op->getBatchSize(), op->getChannels(), 
op->getHeight(), op->getWidth())); } 
MWLeakyReLULayerImpl::~MWLeakyReLULayerImpl() { } void 
MWLeakyReLULayerImpl::predict() { MWTensor* op = getLayer()->getOutputTensor(); 
leakyReLUForwardImpl(op->getHeight(), op->getWidth(), op->getChannels(), 
op->getBatchSize(), oYbqYsqgVhrUzFEKbBbR, getData()); } void 
MWLeakyReLULayerImpl::cleanup() { if (hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(0))); }  if 
(!aLsOwwcceEmRSYzllBNs) { MWTensor* op = getLayer()->getOutputTensor(0); float* 
data = op->getData(); if (data) { call_cuda_free(data); } }  } void __global__ 
leakyReLUImpl(float * AFQBkxwYGKLsACiDKwRM, const double ATEikvMQPqBefhJzjzhc, const int 
CGbFsczkgkhjcHoCKzBx) { int const i = blockDim.x * blockIdx.x + threadIdx.x; if (i < 
CGbFsczkgkhjcHoCKzBx) { float tf = float(AFQBkxwYGKLsACiDKwRM[i]<0); AFQBkxwYGKLsACiDKwRM[i] = 
AFQBkxwYGKLsACiDKwRM[i] - tf*ATEikvMQPqBefhJzjzhc*AFQBkxwYGKLsACiDKwRM[i]; } } void 
leakyReLUForwardImpl(int ZCArwzdUdwQuFQUWjnUE, int vxtNGOWYjhKeBBSzuIMB, int 
jLyhrFjMmVnNjoeDJCwH, int NMMfJylfQjiIUAKhXCJb,  const double 
oYbqYsqgVhrUzFEKbBbR, float* output) { int hljcfGWsvZXJZNrImpJB = 
ZCArwzdUdwQuFQUWjnUE*vxtNGOWYjhKeBBSzuIMB* 
jLyhrFjMmVnNjoeDJCwH*NMMfJylfQjiIUAKhXCJb; int 
sRECVoNNtDdcBOWgDyar = (hljcfGWsvZXJZNrImpJB < 1024) ? hljcfGWsvZXJZNrImpJB : 
1024; int NnAKUXChhnRnQmWsknGy = (hljcfGWsvZXJZNrImpJB + 
sRECVoNNtDdcBOWgDyar - 1)/sRECVoNNtDdcBOWgDyar; 
leakyReLUImpl<<<NnAKUXChhnRnQmWsknGy, sRECVoNNtDdcBOWgDyar>>>( 
output, (1 - oYbqYsqgVhrUzFEKbBbR), hljcfGWsvZXJZNrImpJB); }