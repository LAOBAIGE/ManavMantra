#include "hip/hip_runtime.h"
#include "MWDepthConcatenationLayerImpl.hpp"
#include "MWDepthConcatenationLayer.hpp"
#include <stdarg.h>
#include <cassert>
 MWDepthConcatenationLayerImpl::MWDepthConcatenationLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { 
createDepthConcatenationLayer(); } 
MWDepthConcatenationLayerImpl::~MWDepthConcatenationLayerImpl() {  } void 
MWDepthConcatenationLayerImpl::createDepthConcatenationLayer() {  MWTensor* 
opTensor = getLayer()->getOutputTensor(0);  
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*opTensor->getHeight()*opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(0))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
__global__ concatImpl(float* in, float* out, size_t numElems,  size_t 
batchSize,  size_t outStride,  size_t startOffset)  { size_t i = blockDim.x * 
blockIdx.x + threadIdx.x; size_t maxElems = numElems*batchSize; for (; i < 
maxElems; i += size_t(blockDim.x*gridDim.x)) { size_t batchOffset = i/numElems; 
size_t elemOffset = i - (batchOffset*numElems);  int outOffset = startOffset + 
batchOffset*outStride; out[elemOffset + outOffset] = in[i];  } } void 
MWDepthConcatenationLayerImpl::predict() { int outputOffset = 0; MWTensor* 
opTensor = getLayer()->getOutputTensor(0); int outputStridePerBatch = 
opTensor->getHeight()*opTensor->getWidth()*opTensor->getChannels(); for (int k 
= 0; k < getLayer()->getNumInputs(); k++) { MWTensor* ipTensor = 
getLayer()->getInputTensor(k); int hljcfGWsvZXJZNrImpJB = 
ipTensor->getBatchSize()* ipTensor->getHeight()* ipTensor->getWidth()* 
ipTensor->getChannels();  int sRECVoNNtDdcBOWgDyar = 
ceil(hljcfGWsvZXJZNrImpJB/32)*32; sRECVoNNtDdcBOWgDyar = 
(sRECVoNNtDdcBOWgDyar < 1024) ? sRECVoNNtDdcBOWgDyar : 1024; int 
NnAKUXChhnRnQmWsknGy = (hljcfGWsvZXJZNrImpJB + sRECVoNNtDdcBOWgDyar - 
1)/sRECVoNNtDdcBOWgDyar; int numElemsPerBatch = 
ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels(); 
concatImpl<<<NnAKUXChhnRnQmWsknGy, 
sRECVoNNtDdcBOWgDyar>>>(ipTensor->getData(), getData(), numElemsPerBatch, 
ipTensor->getBatchSize(), outputStridePerBatch, outputOffset); outputOffset += 
numElemsPerBatch; } } void MWDepthConcatenationLayerImpl::cleanup() { if 
(hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } if 
(REXdEoRjxuQJkqgIDihy) { call_cuda_free(REXdEoRjxuQJkqgIDihy); } }