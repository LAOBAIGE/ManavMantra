#include "hip/hip_runtime.h"
#include "MWAdditionLayer.hpp"
#include "MWAdditionLayerImpl.hpp"
#include <stdarg.h>
#include <cassert>
 MWAdditionLayerImpl::MWAdditionLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl)  : MWCNNLayerImpl(layer, ntwk_impl)  { 
createAdditionLayer(); } MWAdditionLayerImpl::~MWAdditionLayerImpl() { } void 
MWAdditionLayerImpl::createAdditionLayer() { MWAdditionLayer* AdditionLayer = 
static_cast<MWAdditionLayer*>(getLayer()); MWTensor* ipTensor = 
AdditionLayer->getInputTensor(0); MWTensor* opTensor = 
AdditionLayer->getOutputTensor(0); 
CUDA_CALL(hipMalloc((void**)&REXdEoRjxuQJkqgIDihy, 
sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels()*ipTensor->getBatchSize())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(0))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(), 
ipTensor->getChannels(), ipTensor->getHeight(), ipTensor->getWidth())); return 
;  } void __global__ addImpl(float* in1, float* in2, float* out, size_t 
maxElems) { size_t i = blockDim.x * blockIdx.x + threadIdx.x; for (; i < 
maxElems; i += size_t(blockDim.x*gridDim.x)) { out[i] = in1[i] + in2[i]; } } 
void MWAdditionLayerImpl::predict() { MWAdditionLayer* AdditionLayer = 
static_cast<MWAdditionLayer*>(getLayer()); MWTensor* ipTensor = 
AdditionLayer->getInputTensor(0); MWTensor* ipTensor1 = 
AdditionLayer->getInputTensor(1); MWTensor* opTensor = 
AdditionLayer->getOutputTensor(0); int hljcfGWsvZXJZNrImpJB = 
ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels()*ipTensor->getBatchSize(); 
int sRECVoNNtDdcBOWgDyar = (hljcfGWsvZXJZNrImpJB < 1024) ? 
hljcfGWsvZXJZNrImpJB : 1024; int NnAKUXChhnRnQmWsknGy = (hljcfGWsvZXJZNrImpJB + 
sRECVoNNtDdcBOWgDyar - 1)/sRECVoNNtDdcBOWgDyar; 
addImpl<<<NnAKUXChhnRnQmWsknGy, sRECVoNNtDdcBOWgDyar>>>( 
ipTensor->getData(), ipTensor1->getData(), getData(), hljcfGWsvZXJZNrImpJB); for 
(int k = 2; k < AdditionLayer->getNumInputs(); k++) { 
addImpl<<<NnAKUXChhnRnQmWsknGy, sRECVoNNtDdcBOWgDyar>>>( 
AdditionLayer->getInputTensor(k)->getData(), getData(), getData(), 
hljcfGWsvZXJZNrImpJB); } } void MWAdditionLayerImpl::cleanup() { MWAdditionLayer* 
AdditionLayer = static_cast<MWAdditionLayer*>(getLayer()); if 
(hasOutputDescriptor()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } for(int idx 
= 0; idx < AdditionLayer->getNumOutputs(); idx++) {  MWTensor* op = 
AdditionLayer->getOutputTensor(idx); float* data = op->getData(); if (data) { 
call_cuda_free(data); } }  }